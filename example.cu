#include "hip/hip_runtime.h"
#include "kernel_float.h"
#include <hip/hip_fp16.h>

namespace kf = kernel_float;

__global__ void kernel(
    kf::vec_ptr<half, 4, const __hip_fp8_e5m2_fnuz> input, 
    float constant, 
    kf::vec_ptr<half, 4> output
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    output(i) = input[i] + kf::cast<half>(constant);
}
