#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <vector>

#include "kernel_float.h"
namespace kf = kernel_float;

void cuda_check(hipError_t code) {
    if (code != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(code));
    }
}

template<int N>
__global__ void my_kernel(
    int length,
    kf::vec_ptr<const half, N> input,
    double constant,
    kf::vec_ptr<half, N, float> output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i * N < length) {
        output(i) = kf::fma(input[i], input[i], kf::cast<__half>(constant));
    }
}

template<int items_per_thread>
void run_kernel(int n) {
    double constant = 1.0;
    std::vector<half> input(n);
    std::vector<float> output_expected;
    std::vector<float> output_result;

    // Generate input data
    for (int i = 0; i < n; i++) {
        input[i] = half(i);
        output_expected[i] = float(i + constant);
    }

    // Allocate device memory
    __half* input_dev;
    float* output_dev;
    cuda_check(hipMalloc(&input_dev, sizeof(half) * n));
    cuda_check(hipMalloc(&output_dev, sizeof(float) * n));

    // Copy device memory
    cuda_check(hipMemcpy(input_dev, input.data(), sizeof(half) * n, hipMemcpyDefault));

    // Launch kernel!
    int block_size = 256;
    int items_per_block = block_size * items_per_thread;
    int grid_size = (n + items_per_block - 1) / items_per_block;
    my_kernel<items_per_thread><<<grid_size, block_size>>>(
        n,
        kf::assert_aligned(input_dev),
        constant,
        kf::assert_aligned(output_dev));

    // Copy results back
    cuda_check(hipMemcpy(output_dev, output_result.data(), sizeof(float) * n, hipMemcpyDefault));

    // Check results
    for (int i = 0; i < n; i++) {
        float result = output_result[i];
        float answer = output_expected[i];

        if (result != answer) {
            std::stringstream msg;
            msg << "error: index " << i << " is incorrect: " << result << " != " << answer;
            throw std::runtime_error(msg.str());
        }
    }

    cuda_check(hipFree(input_dev));
    cuda_check(hipFree(output_dev));
}

int main() {
    int n = 84000;  // divisible by 1, 2, 3, 4, 5, 6, 7, 8
    cuda_check(hipSetDevice(0));

    run_kernel<1>(n);
    run_kernel<2>(n);
    //    run_kernel<3>(n);
    run_kernel<4>(n);
    run_kernel<8>(n);

    std::cout << "result correct\n";
    return EXIT_SUCCESS;
}
